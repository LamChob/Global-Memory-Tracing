#include "TraceUtils.h"
#include "DeviceUtils.h"
#include "hip/hip_runtime.h"

__host__ void trace_start(hipStream_t stream, hipError_t status, void *vargs) {
    //TraceUtils* t = (TraceUtils*) vargs->utils;
    __t->start(((const char*)vargs), stream);
}

__host__ void trace_stop(hipStream_t stream, hipError_t status, void *vargs) {
    //ksdölTraceUtils* t = (TraceUtils*) vargs;
    __t->stop(stream);
}
